#include "hip/hip_runtime.h"
//
// Created by alex on 08.08.24.
//

#include <hip/hip_runtime_api.h>
#include "CudaCommon.cuh"

namespace Bcg::cuda {
    __global__ void kernel() {
        printf("Hello from CUDA Device!\n");
    }

    CudaError HelloFromCudaDevice() {
        kernel<<<1, 1>>>();
        return CudaCheckErrorAndSync();
    }

    CudaError CudaCheckErrorAndSync() {
        // Check for any errors launching the kernel
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            return CudaError::FailedKernelLaunch;
        }

        // Synchronize device
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            //Log::Error("Failed to synchronize: " + func_name + "  " + std::string(hipGetErrorString(err)));
            return CudaError::FailedSynchronisation;
        }
        return CudaError::None;
    }
}