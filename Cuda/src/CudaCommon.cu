#include "hip/hip_runtime.h"
//
// Created by alex on 08.08.24.
//

#include <hip/hip_runtime_api.h>

#include "CudaCommon.cuh"
#include "Logger.h"

namespace Bcg::cuda {
    __global__ void kernel() {
        printf("Hello from CUDA Device!\n");
    }

    void HelloFromCudaDevice() {
        kernel<<<1, 1>>>();
        if (CudaCheckErrorAndSync(__func__)) {
            Log::Info("CUDA kernel " + std::string(__func__) + " executed successfully.");
        }
    }

    bool CudaCheckErrorAndSync(const std::string &func_name) {
        // Check for any errors launching the kernel
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            Log::Error("Failed to launch kernel: " + func_name + " " + std::string(hipGetErrorString(err)));
            return false;
        }

        // Synchronize device
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            Log::Error("Failed to synchronize: " + func_name + "  " + std::string(hipGetErrorString(err)));
            return false;
        }
        return true;
    }
}