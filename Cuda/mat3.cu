#include "hip/hip_runtime.h"
//
// Created by alex on 17.08.24.
//

#include "mat3.cuh"
#include "mat2.cuh"

namespace Bcg {
    __device__ __host__  mat3::mat3() : col0(), col1(), col2() {

    }

    __device__ __host__  mat3::mat3(vec3 col0, vec3 col1, vec3 col2) : col0(col0), col1(col1), col2(col2) {

    }

    __device__ __host__  mat3 mat3::identity() {
        return {{1, 0, 0},
                {0, 1, 0},
                {0, 0, 1}};
    }

    __device__ __host__  mat3 mat3::constant(float c) {
        return {{c, c, c},
                {c, c, c},
                {c, c, c}};
    }

    __device__ __host__  mat3 mat3::reflect_x() {
        return {{-1, 0, 0},
                {0,  1, 0},
                {0,  0, 1}};
    }

    __device__ __host__  mat3 mat3::reflect_y() {
        return {{1, 0,  0},
                {0, -1, 0},
                {0, 0,  1}};
    }

    __device__ __host__  mat3 mat3::reflect_z() {
        return {{1, 0, 0},
                {0, 1, 0},
                {0, 0, -1}};
    }

    __device__ __host__  mat3 mat3::rot(const vec3 axis, float angle) {
        float c = cosf(angle);
        float s = sinf(angle);
        float t = 1 - c;
        vec3 a = axis.normalized();
        return {{t * a.x * a.x + c,       t * a.x * a.y - s * a.z, t * a.x * a.z + s * a.y},
                {t * a.x * a.y + s * a.z, t * a.y * a.y + c,       t * a.y * a.z - s * a.x},
                {t * a.x * a.z - s * a.y, t * a.y * a.z + s * a.x, t * a.z * a.z + c}};
    }

    __device__ __host__ mat3 mat3::scale(const vec3 s){
        return {{s.x, 0, 0},
                {0, s.y, 0},
                {0, 0, s.z}};
    }

    __device__ __host__  mat3 mat3::project(const vec3 &normal) {
        vec3 n = normal.normalized();
        return {{1 - n.x * n.x, -n.x * n.y,    -n.x * n.z},
                {-n.y * n.x,    1 - n.y * n.y, -n.y * n.z},
                {-n.z * n.x,    -n.z * n.y,    1 - n.z * n.z}};
    }


    __device__ __host__  mat3 mat3::shear_x(float s) {
        return {{1, 0, 0},
                {s, 1, 0},
                {0, 0, 1}};
    }

    __device__ __host__  mat3 mat3::shear_y(float s) {
        return {{1, s, 0},
                {0, 1, 0},
                {0, 0, 1}};
    }

    __device__ __host__  mat3 mat3::shear_z(float s) {
        return {{1, 0, s},
                {0, 1, 0},
                {0, 0, 1}};
    }

    __device__ __host__  mat2 mat3::left_upper() const {
        return {{col0.x, col0.y},
                {col1.x, col1.y}};
    }

    __device__ __host__  mat2 mat3::right_upper() const {
        return {{col1.x, col1.y},
                {col2.x, col1.y}};
    }

    __device__ __host__  mat2 mat3::left_lower() const {
        return {{col0.y, col0.z},
                {col1.y, col1.z}};
    }

    __device__ __host__  mat2 mat3::right_lower() const {
        return {{col1.y, col1.z},
                {col2.y, col1.z}};
    }

    __device__ __host__  mat3 mat3::operator-() const {
        return {-col0, -col1, -col2};
    }

    __device__ __host__  vec3 mat3::operator[](int i) const {
        //internally vec3 is a vec4 with the last component set to 0
        return (&col0)[i];
    }

    __device__ __host__  vec3 &mat3::operator[](int i) {
        return (&col0)[i];
    }

    __device__ __host__ const float &mat3::operator()(int r, int c) const{
        return (*this)[c][r];
    }

    __device__ __host__ float &mat3::operator()(int r, int c){
        return (*this)[c][r];
    }

    __device__ __host__  mat3 mat3::operator+(const mat3 &b) const {
        return {col0 + b.col0, col1 + b.col1, col2 + b.col2};
    }

    __device__ __host__  mat3 mat3::operator-(const mat3 &b) const {
        return {col0 - b.col0, col1 - b.col1, col2 - b.col2};
    }

    __device__ __host__  mat3 mat3::operator*(const mat3 &b) const {
        return {col0 * b.col0.x + col1 * b.col0.y + col2 * b.col0.z,
                col0 * b.col1.x + col1 * b.col1.y + col2 * b.col1.z,
                col0 * b.col2.x + col1 * b.col2.y + col2 * b.col2.z};
    }

    __device__ __host__  mat3 mat3::operator+(float b) const {
        return {col0 + b, col1 + b, col2 + b};
    }

    __device__ __host__  mat3 mat3::operator-(float b) const {
        return {col0 - b, col1 - b, col2 - b};
    }

    __device__ __host__  mat3 mat3::operator*(float b) const {
        return {col0 * b, col1 * b, col2 * b};
    }

    __device__ __host__  mat3 mat3::operator/(float b) const {
        return {col0 / b, col1 / b, col2 / b};
    }

    __device__ __host__  vec3 mat3::operator*(const vec3 &v) const {
        return {col0 * v.x + col1 * v.y + col2 * v.z};
    }

    __device__ __host__  mat3 mat3::transpose() const {
        return {{col0.x, col1.x, col2.x},
                {col0.y, col1.y, col2.y},
                {col0.z, col1.z, col2.z}};
    }

    __device__ __host__  float mat3::determinant() const {
        return mat3_determinant(col0.x, col0.y, col0.z, col1.x, col1.y, col1.z, col2.x, col2.y, col2.z);
    }

    __device__ __host__  mat3 mat3::inverse() const {
        return transpose() / determinant();
    }

    __device__ __host__  mat3 mat3::adjoint() const {
        return mat3{
                // First row of cofactors
                vec3{
                        mat2_determinant(col1.y, col1.z, col2.y, col2.z),
                        -mat2_determinant(col1.x, col1.z, col2.x, col2.z),
                        mat2_determinant(col1.x, col1.y, col2.x, col2.y)
                },
                // Second row of cofactors
                vec3{
                        -mat2_determinant(col0.y, col0.z, col2.y, col2.z),
                        mat2_determinant(col0.x, col0.z, col2.x, col2.z),
                        -mat2_determinant(col0.x, col0.y, col2.x, col2.y)
                },
                // Third row of cofactors
                vec3{
                        mat2_determinant(col0.y, col0.z, col1.y, col1.z),
                        -mat2_determinant(col0.x, col0.z, col1.x, col1.z),
                        mat2_determinant(col0.x, col0.y, col1.x, col1.y)
                }
        }.transpose();
    }

    __device__ __host__  mat3 mat3::cofactor() const {
        return adjoint();
    }

    __device__ __host__ mat3 operator+(float a, const mat3 &b) {
        return {a + b.col0, a + b.col1, a + b.col2};
    }

    __device__ __host__ mat3 operator-(float a, const mat3 &b) {
        return {a - b.col0, a - b.col1, a - b.col2};
    }

    __device__ __host__ mat3 operator*(float a, const mat3 &b) {
        return {a * b.col0, a * b.col1, a * b.col2};
    }

    __device__ __host__ mat3 operator/(float a, const mat3 &b) {
        return {a / b.col0, a / b.col1, a / b.col2};
    }

    __device__ __host__ float mat3_determinant(
            float a00, float a01, float a02,
            float a10, float a11, float a12,
            float a20, float a21, float a22) {
        return a00 * mat2_determinant(a11, a12, a21, a22)
               - a01 * mat2_determinant(a10, a12, a20, a22)
               + a02 * mat2_determinant(a10, a11, a20, a21);
    }
}